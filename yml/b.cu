#include "hip/hip_runtime.h"
﻿#include "b.hpp"
#include "common.cuh"

__global__ void hello()
{
    common_hello(2);
}

void b_hello()
{
    hello<<<1, 2>>>();
    hipDeviceSynchronize();
}