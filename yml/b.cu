#include "hip/hip_runtime.h"
﻿#include "b.hpp"
#include "common.cuh"

extern "C" static __global__ void hello()
{
    common_hello(2, threadIdx.x * threadIdx.y + threadIdx.x);
}

void b_hello()
{
    hello<<<1, 2>>>();
    hipDeviceSynchronize();
}