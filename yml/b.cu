#include "hip/hip_runtime.h"
﻿#include "b.hpp"
#include "common.cu"

extern "C" static __global__ void hello()
{
    common_hello(2, threadIdx.x * threadIdx.y + threadIdx.x);
}

void b_hello()
{
    hello<<<4, 4>>>();
    hipDeviceSynchronize();
}