#include "hip/hip_runtime.h"
﻿#include "b.hpp"
#include "common.cuh"

static __global__ void hello()
{
    common_hello(2, threadIdx.x * threadIdx.y + threadIdx.x);
}

void b_hello()
{
    hello<<<4, 4>>>();
    hipDeviceSynchronize();
}