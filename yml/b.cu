#include "hip/hip_runtime.h"
﻿#include "b.hpp"
#include "common.cuh"

static __global__ void hello()
{
    common_hello(2, blockIdx.x * blockDim.x + threadIdx.x);
}

void b_hello()
{
    hello<<<4, 4>>>();
    hipDeviceSynchronize();
}