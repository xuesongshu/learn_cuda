﻿#include "common.cuh"
#include <stdio.h>

__device__ void common_hello(int ichoice)
{
    switch (ichoice)
    {
    case 1:
        printf("greet from A\r\n");
        break;
    case 2:
        printf("greet from B\r\n");
    default:
        break;
    }
}