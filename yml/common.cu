﻿#include "common.cuh"
#include <stdio.h>

extern "C" static __host__ __device__ void common_hello(int ichoice, int ithread_id)
{
    switch (ichoice)
    {
    case 1:
        printf("greet from A, thread%d\r\n", ithread_id);
        break;
    case 2:
        printf("greet from B, thread%d\r\n", ithread_id);
    default:
        break;
    }
}