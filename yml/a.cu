#include "hip/hip_runtime.h"
﻿#include "a.hpp"
#include "common.cuh"

static __global__ void hello()
{
    common_hello(1, blockIdx.x * blockDim.x + threadIdx.x);
}

void a_hello()
{
    hello<<<3, 3>>>();
    hipDeviceSynchronize();
}