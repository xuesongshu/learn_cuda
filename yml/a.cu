#include "hip/hip_runtime.h"
﻿#include "a.hpp"
#include "common.cu"

extern "C" static __global__ void hello()
{
    common_hello(1, threadIdx.x * threadIdx.y + threadIdx.x);
}

void a_hello()
{
    hello<<<1, 2>>>();
    hipDeviceSynchronize();
}