#include "hip/hip_runtime.h"
﻿#include "a.hpp"
#include "common.cuh"

static __global__ void hello()
{
    common_hello(1, threadIdx.x * threadIdx.y + threadIdx.x);
}

void a_hello()
{
    hello<<<3, 3>>>();
    hipDeviceSynchronize();
}