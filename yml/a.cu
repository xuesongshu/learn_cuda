#include "hip/hip_runtime.h"
﻿#include "a.hpp"
#include "common.cuh"

__global__ void hello()
{
    common_hello(1);
}

void a_hello()
{
    hello<<<1, 2>>>();
    hipDeviceSynchronize();
}